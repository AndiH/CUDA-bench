#include "hip/hip_runtime.h"
// #include <thrust/sort.h>
#include <thrust/reduce.h>
// #include <thrust/inner_product.h>
// #include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include <iostream>
// #include <algorithm>
#include <vector>
// #include <tuple>
#include <numeric>
// #include <backward/algo.h>
// #include <boost/timer.hpp>

#include "TH2D.h"
#include "TMatrixD.h"
#include "TROOT.h"
#include "TApplication.h"
#include "TCanvas.h"
#include "TStopwatch.h"
#include "TGraphErrors.h"
#include "TMultiGraph.h"
#include "TLegend.h"
#include "TMath.h"

namespace my {
	template <class ForwardIterator>
	void sequence (ForwardIterator first, ForwardIterator last) {
		int value = 0;
		for (; first != last; ++first) {
			*first = value;
			++value;
		}
	}
	
	template <typename T>
	struct square
	{
		__host__ __device__
		T operator()(const T& x) const { 
			return x * x;
		}
	};
}

int main(int argc, char** argv) {
	int upperBorder = 100000;
	if (argc > 1) upperBorder = atof(argv[1]);
	
	double yOffset = 0; // Needed for log plots -- set to one and uncomment setlogy below
	int incrementNOfNumbersBy = 2; // Can also be changed to + in outer for loop
	int nOfRepetition = 100; // How many times should each data vector be calculated - a mean is taken from those nOfReptition values
	
	std::vector<thrust::tuple<int, double, double, double> > allTheTimes; // nOfNumbers, cpu, gpu_Transfer, gpu_Compute
	std::vector<thrust::tuple<int, double, double, double> > allTheErrors;
	
	for (int nOfNumbers = 10; nOfNumbers <= upperBorder; nOfNumbers = nOfNumbers*incrementNOfNumbersBy) {
		std::vector<double> preAverageTime_cpu;
		std::vector<double> preAverageTime_gpuCopy;
		std::vector<double> preAverageTime_gpuCompute;
		
		thrust::host_vector<int> h_vec(nOfNumbers+1);
		
		// Stuff for timing on GPU
		hipEvent_t start, intermediate, stop;
		hipEventCreate(&start);
		hipEventCreate(&intermediate);
		hipEventCreate(&stop);
		float timeForCopy, timeForComputation;
		
		// Stuff for timing on CPU
		TStopwatch cpuWatch;
		
		
		srand(time(NULL));
		for (int i = 0; i < nOfRepetition; i++) {
			thrust::generate(h_vec.begin(), h_vec.end(), rand); // HOST
			
			// GPU
			hipDeviceSynchronize(); // make sure everything is ready
			
			hipEventRecord(start, 0); // start recording on device
			
			thrust::device_vector<int> d_vec = h_vec; // copy stuff onto device
			
			hipEventRecord(intermediate, 0); // make data point after copy
			
			int d_sumOfSquares = thrust::transform_reduce(d_vec.begin(), d_vec.end(), my::square<int>(), 0, thrust::plus<int>()); // reduce all squared values
			
			hipDeviceSynchronize(); // wait for all threads
			
			hipEventRecord(stop, 0); // stop all counters
			hipEventSynchronize(stop); // wait for stop to finish
			
			hipEventElapsedTime(&timeForCopy, start, intermediate); // time for copy
			hipEventElapsedTime(&timeForComputation, intermediate, stop); // time for computation
			
			
			
			// CPU
			cpuWatch.Start(true); // alternative: "boost::timer myTimer;"
			
			int h_sumOfSquares = thrust::transform_reduce(h_vec.begin(), h_vec.end(), my::square<int>(), 0, thrust::plus<int>()); // runs on host vectors
			
			cpuWatch.Stop();
			
			
			// Fill values
			preAverageTime_cpu.push_back(cpuWatch.CpuTime());
			preAverageTime_gpuCopy.push_back(timeForCopy);
			preAverageTime_gpuCompute.push_back(timeForComputation);
		}
		// Clean up:
		hipEventDestroy(start);
		hipEventDestroy(intermediate);
		hipEventDestroy(stop);
		
		double meanCpu = TMath::Mean(preAverageTime_cpu.size(), &preAverageTime_cpu[0]); // the constructor of Mean using iterators "Mean(bla.begin(), bla.end())" doesn't seem to work
		double meanGpuCopy = TMath::Mean(preAverageTime_gpuCopy.size(), &preAverageTime_gpuCopy[0]);
		double meanGpuCompute = TMath::Mean(preAverageTime_gpuCompute.size(), &preAverageTime_gpuCompute[0]);
		double rmsCpu = TMath::RMS(preAverageTime_cpu.size(), &preAverageTime_cpu[0]);
		double rmsGpuCopy = TMath::RMS(preAverageTime_gpuCopy.size(), &preAverageTime_gpuCopy[0]);
		double rmsGpuCompute = TMath::RMS(preAverageTime_gpuCompute.size(), &preAverageTime_gpuCompute[0]);
		
		allTheTimes.push_back(thrust::make_tuple(nOfNumbers, meanCpu + yOffset, yOffset + meanGpuCopy/1000, yOffset + meanGpuCompute/1000));
		allTheErrors.push_back(thrust::make_tuple(nOfNumbers, rmsCpu + yOffset, yOffset + rmsGpuCopy/1000, yOffset + rmsGpuCompute/1000));
		
		
		std::cout << "Mean Time for " << nOfNumbers << " random numbers * " << nOfRepetition << std::endl;
		std::cout << "  CPU = " << meanCpu << "+-" << rmsCpu << "s" << std::endl;
		std::cout << "  GPU (Copy To) = " << meanGpuCopy/1000 << "+-" << rmsGpuCopy/1000 << "s" << std::endl;
		std::cout << "  GPU (Compute on) = " << meanGpuCompute/1000 << "+-" << rmsGpuCompute/1000 << "s" << std::endl;
	}
	
	TGraphErrors * graphCpu = new TGraphErrors();
	TGraphErrors * graphGpuCopy = new TGraphErrors();
	TGraphErrors * graphGpuCompute = new TGraphErrors();

	for (int i = 0; i < allTheTimes.size(); i++) {
		int nDataPoints = thrust::get<0>(allTheTimes[i]);
		graphCpu->SetPoint(i, nDataPoints, thrust::get<1>(allTheTimes[i]));
		graphGpuCopy->SetPoint(i, nDataPoints, thrust::get<2>(allTheTimes[i]));
		graphGpuCompute->SetPoint(i, nDataPoints, thrust::get<3>(allTheTimes[i]));

		graphCpu->SetPointError(i, 0, thrust::get<1>(allTheErrors[i]));
		graphGpuCopy->SetPointError(i, 0, thrust::get<2>(allTheErrors[i]));
		graphGpuCompute->SetPointError(i, 0, thrust::get<3>(allTheErrors[i]));
	}
	
// 	graphCpu->Print();
// 	graphGpuCompute->Print();
// 	graphGpuCopy->Print(); 
	
	int dotSize = 1;
	graphCpu->SetLineColor(kRed);
	graphCpu->SetFillColor(graphCpu->GetLineColor() - 10);
	graphCpu->SetMarkerStyle(kFullDotLarge);
	graphCpu->SetMarkerSize(dotSize);
	graphCpu->SetMarkerColor(graphCpu->GetLineColor() + 2);
	graphCpu->SetTitle("CPU");
	graphGpuCopy->SetLineColor(kBlue);
	graphGpuCopy->SetFillColor(graphGpuCopy->GetLineColor() - 10);
	graphGpuCopy->SetTitle("GPU - Copy");
	graphGpuCopy->SetMarkerStyle(kFullDotLarge);
	graphGpuCopy->SetMarkerSize(dotSize);
	graphGpuCopy->SetMarkerColor(graphGpuCopy->GetLineColor() + 2);
	graphGpuCompute->SetLineColor(kGreen+2);
	graphGpuCompute->SetFillColor(graphGpuCompute->GetLineColor() - 10);
	graphGpuCompute->SetTitle("GPU - Compute");
	graphGpuCompute->SetMarkerStyle(kFullDotLarge);
	graphGpuCompute->SetMarkerSize(dotSize);
	graphGpuCompute->SetMarkerColor(graphGpuCompute->GetLineColor() + 2);
	
	TApplication *theApp = new TApplication("app", &argc, argv, 0, -1);
	TCanvas * c1 = new TCanvas("c1", "default", 100, 10, 800, 600);
	
	TMultiGraph mg;

	mg.Add(graphCpu);
	mg.Add(graphGpuCopy);
	mg.Add(graphGpuCompute);
	
	mg.Draw("APL");
	mg.GetXaxis()->SetTitle("Random Numbers/#");
	mg.GetYaxis()->SetTitle("Time/s");
	
// 	gPad->SetLogy();
	
	TLegend * leg = c1->BuildLegend(0.1,0.75,0.42,0.9);
	leg->SetFillColor(kWhite);
	graphCpu->Fit("pol1","FQ");
	c1->Update();
	theApp->Run();

}